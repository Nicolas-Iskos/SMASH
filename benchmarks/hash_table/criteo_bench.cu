#include "hip/hip_runtime.h"
#include <SlabHashResizing/src/gpu_hash_table.cuh>
#include <cuco/static_reduction_map.cuh>
#include <single_value_hash_table.cuh>

#include <benchmark/benchmark.h>
#include <synchronization.hpp>
#include <cuco/dynamic_map.cuh>
#include <iostream>
#include <random>

enum class dist_type {
  UNIQUE,
  UNIQUE_NONE,
  UNIFORM,
  GAUSSIAN
};

template<dist_type Dist, typename Key, typename OutputIt>
static void generate_keys(OutputIt output_begin, OutputIt output_end) {
  auto num_keys = std::distance(output_begin, output_end);
  
  std::random_device rd;
  std::mt19937 gen{rd()};

  switch(Dist) {
    case dist_type::UNIQUE:
      for(auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i;
      }
      shuffle(output_begin, output_end, std::default_random_engine(10));
      break;
    case dist_type::UNIQUE_NONE:
      for(auto i = 0; i < num_keys; ++i) {
        output_begin[i] = i + num_keys;
      }
      shuffle(output_begin, output_end, std::default_random_engine(10));
      break;
    case dist_type::UNIFORM:
      for(auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(gen()));
      }
      break;
    case dist_type::GAUSSIAN:
      std::normal_distribution<> dg{1e9, 4e7};
      for(auto i = 0; i < num_keys; ++i) {
        output_begin[i] = std::abs(static_cast<Key>(dg(gen)));
      }
      break;
  }
}

static void gen_final_size(benchmark::internal::Benchmark* b) {
  for(auto size = 10'000'000; size <= 310'000'000; size += 20'000'000) {
    b->Args({size});
  }
}


template <typename Key, typename Value, dist_type Dist>
static void BM_dynamic_insert(::benchmark::State& state) {

  using map_type = cuco::dynamic_map<Key, Value,
                                     cuda::thread_scope_device,
                                     cuco::static_reduction_map>;
  
  std::size_t num_keys = state.range(0);
  std::size_t initial_size = 1<<27;
  
  std::vector<Key> h_keys( num_keys );
  std::vector<cuco::pair_type<Key, Value>> h_pairs ( num_keys );
  
  generate_keys<Dist, Key>(h_keys.begin(), h_keys.end());

  for(auto i = 0; i < num_keys; ++i) {
    Key key = h_keys[i];
    Value val = h_keys[i];
    h_pairs[i].first = key;
    h_pairs[i].second = val;
  }

  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs( h_pairs );

  std::size_t batch_size = 1E7;
  for(auto _ : state) {
    map_type map{initial_size, -1, -1};
    {
      cuda_event_timer raii{state}; 
      for(auto i = 0; i < num_keys; i += batch_size) {
        map.insert(d_pairs.begin() + i, d_pairs.begin() + i + batch_size);
      }
    }
  }

  state.SetBytesProcessed((sizeof(Key) + sizeof(Value)) *
                          int64_t(state.iterations()) *
                          int64_t(state.range(0)));
}