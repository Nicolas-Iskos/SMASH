#include "hip/hip_runtime.h"
#include <cuco/static_reduction_map.cuh>
#include <benchmark/benchmark.h>
#include <synchronization.hpp>
#include <cuco/dynamic_map.cuh>
#include <iostream>
#include <random>
#include <string>
#include <fstream>

using std::string;
using std::fstream;
using std::cout;
using std::endl;
using std::ios;

int num_lines = 195'841'983;
int n_categories = 26;
uint32_t *int_data;

static void gen_category(benchmark::internal::Benchmark* b) {
  for(auto idx = 0; idx < n_categories; ++idx) {
    b->Args({idx});
  }
}

void get_nth_category(string& line, string& output, int n) {
    const int start_offset = 14; // label + 13 non-categorical columns

    int occurrence = 0;
    int index = -1;
    while(occurrence - start_offset < n) {
        if((index = line.find("\t", index + 1)) != string::npos) {
            occurrence++;
        }
    }

    int start_pos = index + 1;
    //cout << "start_pos: " << start_pos << " " << line[start_pos] << endl;
    int end_pos = line.find("\t", start_pos + 1);
    //cout << "end_pos: " << end_pos << endl;

    output = line.substr(start_pos, end_pos - start_pos);
    //output.append(1, '\0');
}

void process_data() {

    fstream criteo_data;
    criteo_data.open("/home/nico/Documents/day_0", ios::in);
    if(criteo_data.is_open()) //checking whether the file is open
    {
       cout << "File successfully opened..." << endl;
       
    }
    else {
        cout << "File not opened!" << endl;
    }

    int_data = new uint32_t[num_lines];
    
    string line;
    for(auto i = 0; i < num_lines; ++i) {
        getline(criteo_data, line);
        string output;
        get_nth_category(line, output, 0);
        int_data[i] = stol(output, NULL, 16);
    }

    cout << std::hex << int_data[0] << " " << int_data[1] << " " << int_data[2] << endl;    
  
    criteo_data.close();    //close the file object
}

template <typename Key, typename Value>
static void BM_dynamic_insert(::benchmark::State& state) {

  using map_type = cuco::dynamic_map<cuco::reduce_add<Value>,
                                     Key, Value,
                                     cuda::thread_scope_device,
                                     cuco::cuda_allocator<char>,
                                     cuco::static_reduction_map>;
  
  int category = state.range(0);
  std::size_t num_keys = num_lines;

  // initial size 4 MB
  std::size_t initial_size = 1<<22;
  
  std::vector<Key> h_keys( num_keys );

  fstream criteo_data;
  criteo_data.open("/home/nico/Documents/category_" + std::to_string(category));
  if(criteo_data.is_open()) {
    cout << "File successfully opened..." << endl;
  }
  else {
    cout << "File not opened!" << endl;
  }

  // read file data into key buffer
  for(auto i = 0; i < num_keys; ++i) {
    string line;
    getline(criteo_data, line);
    h_keys[i] = stol(line, NULL, 16);
  }
  criteo_data.close();

  std::vector<cuco::pair_type<Key, Value>> h_pairs ( num_keys );
  
  for(auto i = 0; i < num_keys; ++i) {
    Key key = h_keys[i];
    Value val = h_keys[i];
    h_pairs[i].first = key;
    h_pairs[i].second = 1;
  }

  thrust::device_vector<cuco::pair_type<Key, Value>> d_pairs( h_pairs );

  std::size_t batch_size = 1E6;
  for(auto _ : state) {
    map_type map{initial_size, -1, -1};
    {
      cuda_event_timer raii{state};
      std::size_t num_remaining = num_keys;
      std::size_t insert_size = 0;
      for(auto i = 0; i < num_keys; i += insert_size) {
        insert_size = min(batch_size, num_remaining);
        map.insert(d_pairs.begin() + i, d_pairs.begin() + i + insert_size);
        num_remaining -= insert_size;
      }
    }

    std::cout << "map size: " << map.get_size() << std::endl;
  }

  state.SetBytesProcessed((sizeof(Key) + sizeof(Value)) *
                          int64_t(state.iterations()) *
                          int64_t(state.range(0)));
}

BENCHMARK_TEMPLATE(BM_dynamic_insert, int32_t, int32_t)
  ->Unit(benchmark::kMillisecond)
  ->Apply(gen_category)
  ->UseManualTime();